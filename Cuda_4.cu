#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define BASE_TYPE double

__global__ void matrixMult(const BASE_TYPE *A, const BASE_TYPE *B, BASE_TYPE *C, int Acols, int Bcols)
{
    int i0 = Acols * (blockDim.y * blockIdx.y + threadIdx.y);
    int j0 = blockDim.x * blockIdx.x + threadIdx.x;
    BASE_TYPE sum = 0;
    for (int k = 0; k < Acols; k++)
        sum += A[i0 + k] * B[k * Bcols + j0];
    int ind = Bcols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

__global__ void matrixMultShared(const BASE_TYPE *A, const BASE_TYPE *B, BASE_TYPE *C, int Acols, int Bcols)
{
    // индекс начала первой подматрицы А, которую обрабатывает блок
    int aBegin = Acols * blockDim.y * blockIdx.y;
    
    // индекс конца подматрицы А, которую обрабатывает блок
    int aEnd = aBegin + Acols - 1;
    
    // шаг для перебора подматриц А
    int aStep = blockDim.x;
    
    // индекс начала первой подматрицы В, которую обрабатывает блок
    int bBegin = blockDim.x * blockIdx.x;
    
    // шаг для перебора подматриц В
    int bStep = blockDim.y * Bcols;
    
    // Выделение разделяемой памяти для подматриц
    __shared__ BASE_TYPE as[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ BASE_TYPE bs[BLOCK_SIZE][BLOCK_SIZE];
    
    // переменная для вычисления элемента подматрицы
    BASE_TYPE sum = 0.0;

    for (int ia = aBegin, ib = bBegin; ia < aEnd; ia += aStep, ib += bStep)
    {
        // загрузка подматриц А и В из глобальной памяти в разделяемую
        as[threadIdx.y][threadIdx.x] = A[ia + Acols * threadIdx.y + threadIdx.x];
        bs[threadIdx.y][threadIdx.x] = B[ib + Bcols * threadIdx.y + threadIdx.x];
        
        // синхронизация нитей
        __syncthreads();
        
        // перемножение двух матриц
        for (int k = 0; k < blockDim.x; k++)
            sum += as[threadIdx.y][k] * bs[k][threadIdx.x];
        
        // синхронизация нитей
        __syncthreads();
    }
    
    // индекс результирующего элемента в глобальной памяти
    int ind = Bcols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    
    // запись элемента в глобальную память
    C[ind] = sum;
}


int toMultiple(int a, int b)
{
    int mod = a % b;
    if (mod != 0)
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}

int main()
{
    hipEvent_t start, stop, shared_start, shared_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&shared_start);
    hipEventCreate(&shared_stop);

    int Arows = 1000;
    int Acols = 2000;
    int Brows = Acols;
    int Bcols = 1500;

    Arows = toMultiple(Arows, BLOCK_SIZE);
    Acols = toMultiple(Acols, BLOCK_SIZE);
    Brows = toMultiple(Brows, BLOCK_SIZE);
    Bcols = toMultiple(Bcols, BLOCK_SIZE);

    size_t Asize = Arows * Acols * sizeof(BASE_TYPE);
    size_t Bsize = Brows * Bcols * sizeof(BASE_TYPE);
    size_t Csize = Arows * Bcols * sizeof(BASE_TYPE);

    BASE_TYPE *h_A = (BASE_TYPE *)malloc(Asize);
    BASE_TYPE *h_B = (BASE_TYPE *)malloc(Bsize);

    for (int i = 0; i < Arows * Acols; ++i)
        h_A[i] = rand() / (BASE_TYPE)RAND_MAX;
    for (int i = 0; i < Brows * Bcols; ++i)
        h_B[i] = rand() / (BASE_TYPE)RAND_MAX;

    BASE_TYPE *d_A = NULL;
    hipMalloc((void **)&d_A, Asize);

    BASE_TYPE *d_B = NULL;
    hipMalloc((void **)&d_B, Bsize);

    BASE_TYPE *d_C = NULL;
    hipMalloc((void **)&d_C, Csize);

    BASE_TYPE *d_C_shared = NULL;
    hipMalloc((void **)&d_C_shared, Csize);

    hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

    // параметры запуска ядра
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid = dim3(Bcols / BLOCK_SIZE, Arows / BLOCK_SIZE);

    hipEventRecord(start, 0);
    matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, Acols, Bcols);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventRecord(shared_start, 0);
    matrixMultShared<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C_shared, Acols, Bcols);
    hipEventRecord(shared_stop, 0);
    hipEventSynchronize(shared_stop);

    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", kernelTime);

    hipEventElapsedTime(&kernelTime, shared_start, shared_stop);
    printf("Shared KernelTime: %.2f milliseconds\n", kernelTime);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_shared);

    free(h_A);
    free(h_B);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(shared_start);
    hipEventDestroy(shared_stop);

    return 0;
}