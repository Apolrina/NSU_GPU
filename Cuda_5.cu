#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#define BLOCK_SIZE 16
#define BASE_TYPE double

int toMultiple(int a, int b)
{
    int mod = a % b;
    if (mod != 0)
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int Arows = 1000;
    int Acols = 2000;
    int Brows = Acols;
    int Bcols = 1500;

    Arows = toMultiple(Arows, BLOCK_SIZE);
    printf("Arows = %d\n", Arows);

    Acols = toMultiple(Acols, BLOCK_SIZE);
    printf("Acols = %d\n", Acols);

    Brows = toMultiple(Brows, BLOCK_SIZE);
    printf("Brows = %d\n", Brows);

    Bcols = toMultiple(Bcols, BLOCK_SIZE);
    printf("Bcols = %d\n", Bcols);

    size_t Asize = Arows * Acols * sizeof(BASE_TYPE);
    size_t Bsize = Brows * Bcols * sizeof(BASE_TYPE);
    size_t Csize = Arows * Bcols * sizeof(BASE_TYPE);

    BASE_TYPE *h_A = (BASE_TYPE *)malloc(Asize);
    BASE_TYPE *h_B = (BASE_TYPE *)malloc(Bsize);
    BASE_TYPE *h_C = (BASE_TYPE *)malloc(Csize);


    for (int i = 0; i < Arows * Acols; ++i)
        h_A[i] = rand() / (BASE_TYPE)RAND_MAX;
    for (int i = 0; i < Brows * Bcols; ++i)
        h_B[i] = rand() / (BASE_TYPE)RAND_MAX;

    BASE_TYPE *h_A_col = (BASE_TYPE *)malloc(Asize);
    BASE_TYPE *h_B_col = (BASE_TYPE *)malloc(Bsize);
    BASE_TYPE *h_C_col = (BASE_TYPE *)malloc(Csize);


    for (int i = 0; i < Arows; i++) {
        for (int j = 0; j < Acols; j++) {
            h_A_col[j * Arows + i] = h_A[i * Acols + j];
        }
    }

    
    for (int i = 0; i < Brows; i++) {
        for (int j = 0; j < Bcols; j++) {
            h_B_col[j * Brows + i] = h_B[i * Bcols + j];
        }
    }

    BASE_TYPE *d_A = NULL;
    hipMalloc((void **)&d_A, Asize);

    BASE_TYPE *d_B = NULL;
    hipMalloc((void **)&d_B, Bsize);

    BASE_TYPE *d_C = NULL;
    hipMalloc((void **)&d_C, Csize);

    hipMemcpy(d_A, h_A_col, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_col, Bsize, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const double alpha = 1.0;
    const double beta = 0.0;

    hipEventRecord(start, 0);
    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                Arows, Bcols, Acols,
                &alpha,
                d_A, Arows,
                d_B, Brows,
                &beta,
                d_C, Arows);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", KernelTime);

    hipMemcpy(h_C_col, d_C, Csize, hipMemcpyDeviceToHost);

    for (int i = 0; i < Arows; i++) {
        for (int j = 0; j < Bcols; j++) {
            h_C[i * Bcols + j] = h_C_col[j * Arows + i];
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_A_col);
    free(h_B_col);
    free(h_C_col);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(handle);

    return 0;
}